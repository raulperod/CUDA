#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define THREADS_PER_BLOCK 512

__global__ void suma( int *a, int *b, int *c ) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index]
}

void llenarMatriz(int*);

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c; 
    int size = N * N * sizeof( int ); 
    
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, size );
    a = (int*)malloc( size );
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    
    llenarMatriz( a );
    llenarMatriz( b );
    
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
    
    suma<<< (N*N+1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
    
    hipMemcpy( c, dev_c, size , hipMemcpyDeviceToHost );
    
    free( a ); 
    free( b ); 
    free( c );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}

void llenarMatriz(int *m){
    for(int i=0 ; i < N ; i++){ 
        for(int j=0 ; j < N ; j++){
            m[j+i*N] = 1; 
        }
    }
}
