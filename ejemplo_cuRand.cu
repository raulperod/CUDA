#include "hip/hip_runtime.h"
/*
Primero hay que cargar bien las librerías

  export LD_LIBRARY_PATH=/usr/local/cuda-8.0/lib64

luego, compilamos

  nvcc ejemplo.cu -o ejemplo -lcurand

*/


/* This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int main(int argc, char *argv[])
{
  size_t n = 100;
  size_t i;
  hiprandGenerator_t gen;
  float *devData, *hostData;
  /* Allocate n floats on host */
  hostData = (float *)malloc(n*sizeof(float));

  /* Allocate n floats on device */
     hipMalloc((void **)&devData, n * sizeof(float));

  /* Create pseudo-random number generator */
     hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  /* Set seed */
     hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)clock());
  /* Generate n floats on device */
     hiprandGenerateUniform(gen, devData, n);
  /* Copy device memory to host */
     hipMemcpy(hostData, devData, n * sizeof(float),
                       hipMemcpyDeviceToHost);
  /* Set seed */
  //   hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

  /* Generate n floats on device */
  //   hiprandGenerateUniform(gen, devData, n);

  /* Copy device memory to host */
  //   hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);

  /* Show result */
  for(i = 0; i < n; i++) {
    printf("%1.4f \n", hostData[i]);
  }
  printf("\n");
  /* Cleanup */
  hiprandDestroyGenerator(gen);
  hipFree(devData);
  free(hostData);
  return 0;
}
