#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 2048

__global__ void suma( int *a, int *b, int *c ) {
    int index = blockIdx.x + blockIdx.y * blockDim.x;
    c[index] = a[index] + b[index]
}

void llenarMatriz(int*);

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c; 
    int size = N * N * sizeof( int ); 
    
    hipMalloc( &dev_a, size );
    hipMalloc( &dev_b, size );
    hipMalloc( &dev_c, size );
    a = (int*)malloc( size );
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    
    llenarMatriz( a );
    llenarMatriz( b );
    
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
    
    dim3 numeroHilos(1,1);
    dim3 numeroBloques(2048,2048);

    suma<<< numeroBloques, numeroHilos  >>>( dev_a, dev_b, dev_c );
    
    hipMemcpy( c, dev_c, size , hipMemcpyDeviceToHost );
    
    free( a ); free( b ); free( c );
    hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
    return 0;
}

void llenarMatriz(int *m){
    for(int i=0 ; i < N ; i++){ 
        for(int j=0 ; j < N ; j++){
            m[j+i*N] = 1; 
        }
    }
}
