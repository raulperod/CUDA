
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// curand
#define N 100
#define T 4

__global__ void multiplicacion( int *a, int *b, int *c ) {
    int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 2047
    int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 2047
        
    c[j+i*N] = 0; // 4,194,303

    for(int k=0 ; k < N ; k++ ){
        c[j+i*N] += a[k+i*N] * b[j+k*N];
    }
}

void llenarMatriz(int*);

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c; 
    int size = N * N * sizeof( int ); 
    // crear evento
    hipEvent_t start, end;
    hipEventCreate(&start); 
    hipEventCreate(&end);

    hipMalloc( &dev_a, size );
    hipMalloc( &dev_b, size );
    hipMalloc( &dev_c, size );
    a = (int*)malloc( size );
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    
    llenarMatriz( a );
    llenarMatriz( b );
    
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
    
    dim3 numeroHilos(T, T);
    dim3 numeroBloques(N/T, N/T);
    // --------------------
    hipEventRecord(start);
    multiplicacion<<< numeroHilos, numeroBloques  >>>( dev_a, dev_b, dev_c );
    hipEventRecord(end);
    //---------------------
    hipMemcpy( c, dev_c, size , hipMemcpyDeviceToHost );
    // sincronizar
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("c[0][0] = %d , c[99][99] = %d\n", c[0], c[N*N-1] );
    printf("Tiempo: %f\n", milliseconds);

    free( a ); free( b ); free( c );
    hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
    return 0;
}

void llenarMatriz(int *m){
    for(int i=0 ; i < N ; i++){ 
        for(int j=0 ; j < N ; j++){
            m[j+i*N] = 1; 
        }
    }
}