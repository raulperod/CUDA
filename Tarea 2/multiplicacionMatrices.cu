
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define T 4

__global__ void multiplicacion( int *a, int *b, int *c ) {
    int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 2047
    int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 2047
        
    c[j+i*N] = 0; // 4,194,303
    for(int k=0 ; k < N ; k++ ){
        c[j+i*N] += a[k+i*N] * b[j+k*N];
    }
}

void llenarMatriz(int*);

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c; 
    int size = N * N * sizeof( int ); 
    
    hipMalloc( &dev_a, size );
    hipMalloc( &dev_b, size );
    hipMalloc( &dev_c, size );
    a = (int*)malloc( size );
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    
    llenarMatriz( a );
    llenarMatriz( b );
    
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
    
    dim3 numeroHilos(T, T);
    dim3 numeroBloques(N/T, N/T);

    multiplicacion<<< numeroBloques, numeroHilos  >>>( dev_a, dev_b, dev_c );
    
    hipMemcpy( c, dev_c, size , hipMemcpyDeviceToHost );
    
    free( a ); free( b ); free( c );
    hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
    return 0;
}

void llenarMatriz(int *m){
    for(int i=0 ; i < N ; i++){ 
        for(int j=0 ; j < N ; j++){
            m[j+i*N] = 1; 
        }
    }
}