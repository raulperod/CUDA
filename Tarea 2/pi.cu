
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
// curand
#define N_TOTAL 4194304
#define N 2048
#define T 32

__global__ void aproximarPi( float *x, float *y, int *z ) {
    int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 2047
    int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 2047
    int index = j + i*N; // 0 - 4194303     
    
    if( (x[index] * x[index] + y[index] * y[index]) <= 1.0f){
        atomicAdd(z, 1);
    }

}

void llenarRandom(float *a, float *b);

int main( void ) {
    float *x, *y, pi; 
    int *z;
    float *dev_x, *dev_y;
    int *dev_z; 
    int size = N_TOTAL * sizeof( float );
    
    hipEvent_t start, end;
    hipEventCreate(&start); 
    hipEventCreate(&end);
    
    hipMalloc( &dev_x, size );
    hipMalloc( &dev_y, size );
    hipMalloc( &dev_z, sizeof( int ) );

    x = (float*)malloc( size );
    y = (float*)malloc( size );
    z = (int*)malloc( sizeof( int ) );
    
    llenarRandom( x, y );
    *z = 0;
    
    hipMemcpy( dev_x, x, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_y, y, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_z, z, sizeof( int ) , hipMemcpyHostToDevice );
    
    dim3 numeroHilos(T, T);
    dim3 numeroBloques(N/T, N/T);
    // --------------------
    hipEventRecord(start);
    aproximarPi<<< numeroBloques, numeroHilos  >>>( dev_x, dev_y, dev_z );
    hipEventRecord(end);
    //---------------------
    hipMemcpy( z, dev_z, sizeof( int ) , hipMemcpyDeviceToHost );
    // sincronizar
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    pi = ((4.0 * (*z)) / N_TOTAL);
    printf("z: %d\n", *z );
    printf("Pi: %f\n", pi );
    printf("Tiempo: %f\n", milliseconds);

    free( x ); free( y ); free( z );
    hipFree( dev_x ); hipFree( dev_y ); hipFree( dev_z );
    return 0;
}

void llenarRandom(float *a, float *b){
    float *v1, *v2;
    v1 = a;
    v2 = b;
    srand(time(NULL));
    for(int i = 0 ; i < N_TOTAL ; i++ ) {
        v1[i] = rand() / (RAND_MAX + 1.0f);
        v2[i] = rand() / (RAND_MAX + 1.0f);
    }
}