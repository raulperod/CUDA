#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
// hiprand
#define N_TOTAL 1048576
#define N 1024
#define T 512

__global__ void aproximarPi( int *x, int *y, int *z ) {
    int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 1023
    int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 1023
    int index = j + i*N; // 0 - 1048576     
    
    if( (x[index] * x[index] + y[index] * y[index]) <= 1.0f){
        atomicAdd( z , 1 );
    }

}

void llenarRandom(float*);

int main( void ) {
    float *x, *y, *z;
    float *dev_x, *dev_y, *dev_z; 
    int size = N_TOTAL * sizeof( float );
    
    hipEvent_t start, end;
    hipEventCreate(&start); 
    hipEventCreate(&end);
    
    hipMalloc( &dev_x, size );
    hipMalloc( &dev_y, size );
    hipMalloc( &dev_z, sizeof( int ) );

    x = (float*)malloc( size );
    y = (float*)malloc( size );
    z = (int*)malloc( sizeof( int ) )
    
    llenarRandom( x );
    llenarRandom( y );
    
    hipMemcpy( dev_x, x, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_y, y, size, hipMemcpyHostToDevice );
    
    dim3 numeroHilos(T, T);
    dim3 numeroBloques(N/T, N/T);
    // --------------------
    hipEventRecord(start);
    aproximarPi<<< numeroHilos, numeroBloques  >>>( dev_x, dev_y, dev_z );
    hipEventRecord(end);
    //---------------------
    hipMemcpy( z, dev_z, sizeof( int ) , hipMemcpyDeviceToHost );
    // sincronizar
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("Pi: %f\n", 4.0 * z / N_TOTAL );
    printf("Tiempo: %f\n", milliseconds);

    free( a ); free( b ); free( c );
    hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
    return 0;
}

void llenarRandom(float *v){
    srand(time(NULL));
    for(int i = 0 ; i < N ; i++ ) {
        v[i] = rand() / (RAND_MAX + 1.0f);
    }
}